#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>

#define MIN_VALUE (-1e38)
typedef float F;

__global__ void kernel_forward(const int B, const int T, const int C,
                               const F *__restrict__ const _w, const F *__restrict__ const _u, const F *__restrict__ const _k, const F *__restrict__ const _v,
                               F *__restrict__ const _y) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int _b = idx / C;
    const int _c = idx % C;
    const int _offset = _b * T * C + _c;

    F u = _u[_c];
    F w = -exp(_w[_c]);
    const F *__restrict__ const k = _k + _offset;
    const F *__restrict__ const v = _v + _offset;
    F *__restrict__ const y = _y + _offset;

    // aa and bb are running sums divided by exp(pp) (to avoid overflow)
    F aa = 0, bb = 0, pp = MIN_VALUE;
    for (int i = 0; i < T; i++) {
        const int ii = i * C;
        const F kk = k[ii];
        const F vv = v[ii];

        F ww = u + kk;
        F p = max(pp, ww);
        F e1 = exp(pp - p);
        F e2 = exp(ww - p);
        y[ii] = (e1 * aa + e2 * vv) / (e1 * bb + e2);
        
        ww = w + pp;
        p = max(ww, kk);
        e1 = exp(ww - p);
        e2 = exp(kk - p);
        aa = e1 * aa + e2 * vv;
        bb = e1 * bb + e2;
        pp = p;
    }
}


void cuda_forward(int B, int T, int C, float *w, float *u, float *k, float *v, float *y) {
    dim3 threadsPerBlock( min(C, 32) ); // requires --maxrregcount 60 for optimal performance
    assert(B * C % threadsPerBlock.x == 0);
    dim3 numBlocks(B * C / threadsPerBlock.x);
    kernel_forward<<<numBlocks, threadsPerBlock>>>(B, T, C, w, u, k, v, y);
}
